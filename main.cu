#include <SDL.h>
#include <SDL_image.h>
#include <SDL_ttf.h>
#include <SDL_mixer.h>
#include <iostream>
#include <stdlib.h>  
#include <crtdbg.h>   //for malloc and free
#include <set>
#include <vector>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define _CRTDBG_MAP_ALLOC
#ifdef _DEBUG
#define new new( _NORMAL_BLOCK, __FILE__, __LINE__)
#endif

SDL_Window* window;
SDL_Renderer* renderer;
bool running;
SDL_Event event;
std::set<std::string> keys;
std::set<std::string> currentKeys;
int mouseX = 0;
int mouseY = 0;
int mouseDeltaX = 0;
int mouseDeltaY = 0;
int mouseScroll = 0;
std::set<int> buttons;
std::set<int> currentButtons;
const int WIDTH = 600;
const int HEIGHT = 600;


int exponent = 4;
int variables = 5;
int mod = 5;

void debug(int line, std::string file) {
	std::cout << "Line " << line << " in file " << file << ": " << SDL_GetError() << std::endl;
}

int factorial(int n) {
	int result = 1;
	for (int i = n; i > 0; i--) {
		result *= i;
	}
	return result;
}

int multinomial(int n, std::vector<int> r) {
	int result = factorial(n);
	for (int i : r) {
		result /= factorial(i);
	}
	return result % mod;
}


std::vector<int> sequenceGenerator(int start, int end, std::vector<int>* subsequence, std::vector<int> count, int variable) {
	std::vector<int> result;
	if (variable == 1) {
		subsequence = new std::vector<int>();
		for (int i = end - start; i >= 0; i--) {
			subsequence->clear();
			for (int j = 0; j < i; j++) {
				subsequence->push_back(variable);
			}
			count.push_back(i);
			if (start + i < end) {
				std::vector<int> a = sequenceGenerator(start + i, end, subsequence, count, variable + 1);
				result.insert(result.end(), a.begin(), a.end());
			}
			else {
				int m = multinomial(end, count);
				for (int i = 0; i < m; i++) {
					result.insert(result.end(), subsequence->begin(), subsequence->end());
				}
			}
			count.pop_back();
		}
		delete subsequence;
	}
	else if (variable < variables) {
		std::vector<int>* subsubsequence = new std::vector<int>();
		for (int i = end - start; i >= 0; i--) {
			subsubsequence->clear();
			subsubsequence->insert(subsubsequence->end(), subsequence->begin(), subsequence->end());
			for (int j = 0; j < i; j++) {
				subsubsequence->push_back(variable);
			}
			count.push_back(i);
			if (i + start < end) {
				std::vector<int> a = sequenceGenerator(start + i, end, subsubsequence, count, variable + 1);
				result.insert(result.end(), a.begin(), a.end());
			}
			else {
				int m = multinomial(end, count);
				for (int i = 0; i < m; i++) {
					result.insert(result.end(), subsubsequence->begin(), subsubsequence->end());
				}
			}
			count.pop_back();
		}
		delete subsubsequence;
	}
	else if (variable == variables) {
		for (int i = end - start; i > 0; i--) {
			subsequence->push_back(variable);
		}
		count.push_back(end - start);
		int m = multinomial(end, count);
		for (int i = 0; i < m; i++) {
			result.insert(result.end(), subsequence->begin(), subsequence->end());
		}
		count.pop_back();
	}
	return result;
}

const Uint32 red = 0x01000000, green = 0x00010000, blue = 0x00000100;
const Uint32 colors[] = {
	0,
	255 * (red + blue + green),
	255 * red, 
	255 * green, 
	255 * blue, 
	255 * red + 255 * green, 
	255 * red + 255 * blue, 
	255 * green + 255 * blue,
	128 * red,
	128 * green,
	128 * blue,
	128 * (red + green),
	128 * (red + blue),
	128 * (green + blue),
	153 * (red + green) + 255 * blue,
	153 * red + 51 * green + 102 * blue,
	102 * (red + blue),
	255 * red + 128 * (green + blue)
};

int main(int argc, char* argv[]) {
	if (SDL_Init(SDL_INIT_EVERYTHING) == 0 && TTF_Init() == 0 && Mix_OpenAudio(44100, MIX_DEFAULT_FORMAT, 2, 2048) == 0) {
		//Setup
		window = SDL_CreateWindow("Weaves", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH, HEIGHT, 0);
		if (window == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		renderer = SDL_CreateRenderer(window, -1, 0);
		if (renderer == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}
		void* txtPixels;
		int pitch;
		Uint32* pixel_ptr;

		Uint8 c;

		std::vector<int> sequence = sequenceGenerator(0, exponent, NULL, {}, 1);

		SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888,
			SDL_TEXTUREACCESS_STREAMING, sequence.size(), sequence.size());
		SDL_LockTexture(texture, NULL, &txtPixels, &pitch);
		pixel_ptr = (Uint32*)txtPixels;
		for (int i = 0; i < sequence.size() * sequence.size(); i++) {
			c = 0;
			c += sequence[i % sequence.size()];
			c += sequence[i / sequence.size()];
			pixel_ptr[i] = colors[c - 2] + 255;
		}
		SDL_UnlockTexture(texture);
		SDL_RenderCopy(renderer, texture, NULL, NULL);
		SDL_RenderPresent(renderer);

		//Main loop
		running = true;
		while (running) {
			//handle events
			for (std::string i : keys) {
				currentKeys.erase(i); //make sure only newly pressed keys are in currentKeys
			}
			for (int i : buttons) {
				currentButtons.erase(i); //make sure only newly pressed buttons are in currentButtons
			}
			mouseScroll = 0;
			while (SDL_PollEvent(&event)) {
				switch (event.type) {
				case SDL_QUIT:
					running = false;
					break;
				case SDL_KEYDOWN:
					if (!keys.contains(std::string(SDL_GetKeyName(event.key.keysym.sym)))) {
						currentKeys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym)));
					}
					keys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym))); //add keydown to keys set
					break;
				case SDL_KEYUP:
					keys.erase(std::string(SDL_GetKeyName(event.key.keysym.sym))); //remove keyup from keys set
					break;
				case SDL_MOUSEMOTION:
					mouseX = event.motion.x;
					mouseY = event.motion.y;
					mouseDeltaX = event.motion.xrel;
					mouseDeltaY = event.motion.yrel;
					break;
				case SDL_MOUSEBUTTONDOWN:
					if (!buttons.contains(event.button.button)) {
						currentButtons.insert(event.button.button);
					}
					buttons.insert(event.button.button);
					break;
				case SDL_MOUSEBUTTONUP:
					buttons.erase(event.button.button);
					break;
				case SDL_MOUSEWHEEL:
					mouseScroll = event.wheel.y;
					break;
				}
			}
		}

		SDL_DestroyTexture(texture);

		//Clean up
		if (window) {
			SDL_DestroyWindow(window);
		}
		if (renderer) {
			SDL_DestroyRenderer(renderer);
		}
		TTF_Quit();
		Mix_Quit();
		IMG_Quit();
		SDL_Quit();
		return 0;
	}
	else {
		return 0;
	}
}